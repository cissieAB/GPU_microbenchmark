//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the maximum read bandwidth of L1 cache for 32 bit read

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 1024
#define WARP_SIZE 32
#define L1_SIZE 32768

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	
	// a register to avoid compiler optimization
	float sink0 = 0;
	float sink1 = 0;
	float sink2 = 0;
	float sink3 = 0;

	for (uint32_t i = 4*tid; i<L1_SIZE; i+=THREADS_NUM*4) {
		float* ptr = posArray + i;
		asm volatile ("{\t\n"
			".reg .f32 data<4>;\n\t"
			"ld.global.ca.v4.f32 {data0,data1,data2,data3}, [%4];\n\t"
			"add.f32 %0, data0, %0;\n\t"
			"add.f32 %0, data1, %1;\n\t"
			"add.f32 %0, data2, %2;\n\t"
			"add.f32 %0, data3, %3;\n\t"	
			"}" : "+f"(sink0),"+f"(sink1),"+f"(sink2),"+f"(sink3) : "l"(ptr) : "memory"
		);
	}
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l1 cache and accumulate
	for(uint32_t j=0; j<(L1_SIZE/2); j++){
        	for (uint32_t i = 4*tid; i<(L1_SIZE/2); i+=(THREADS_NUM*4)){
        	        float* ptr = posArray + i + j;
	                asm volatile ("{\t\n"
				".reg .f32 data<4>;\n\t"
        	                "ld.global.ca.v4.f32 {data0,data1,data2,data3}, [%4];\n\t"
	                        "add.f32 %0, data0, %0;\n\t"
                        	"add.f32 %0, data1, %1;\n\t"
                	        "add.f32 %0, data2, %2;\n\t"
       		                "add.f32 %0, data3, %3;\n\t"
	                        "}" : "+f"(sink0),"+f"(sink1),"+f"(sink2),"+f"(sink3) : "l"(ptr) : "memory"
                	);
        	}
	}
        // stop timing
        //uint32_t stop = 0;
        //asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	
	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink0+sink1+sink2+sink3;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	float *posArray = (float*) malloc(L1_SIZE*sizeof(float));
	float *dsink = (float*) malloc(THREADS_NUM*sizeof(float));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        float *posArray_g;
        float *dsink_g;
	
	for (uint32_t i=0; i<L1_SIZE; i++)
		posArray[i] = (float)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, L1_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(float)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, L1_SIZE*sizeof(float), hipMemcpyHostToDevice) );


	l1_bw<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);
        gpuErrchk( hipPeekAtLastError() );
	
	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(float), hipMemcpyDeviceToHost) );
/*
	for(uint32_t i=0; i<256; i++){
		printf("stop Clk(%d) = %u    \n", i, stopClk);
		printf("start Clk(%d) = %u    \n", i, startClk);
		printf("Clk(%d) = %u \n", i, stopClk-startClk);
		//printf("dsink(%d) = %f \n", i, dsink);
	}
*/
	double bw;
	bw = (double)(L1_SIZE*L1_SIZE/4*4)/((double)(stopClk[0]-startClk[0]));
	printf("L1 bandwidth = %f (byte/clk)\n", bw);
        printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
}