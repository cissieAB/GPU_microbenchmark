//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the maximum read bandwidth of L1 cache for 64 bit read

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 512
#define WARP_SIZE 32
#define L1_SIZE 16384

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, double *dsink, double *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	
	// a register to avoid compiler optimization
	double sink = 0;

//for (uint32_t j = 0; j<32; j++){	
	// populate l1 cache to warm up
	for (uint32_t i = tid; i<L1_SIZE; i+=THREADS_NUM) {
		double* ptr = posArray + i;
		asm volatile ("{\t\n"
			".reg .f64 data;\n\t"
			"ld.global.ca.f64 data, [%1];\n\t"
			"add.f64 %0, data, %0;\n\t"
			"}" : "+d"(sink) : "l"(ptr) : "memory"
		);
	}
//}
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l1 cache and accumulate
	
	for (uint32_t j = 0; j<(L1_SIZE/2); j++){
	        for (uint32_t i = tid; i<(L1_SIZE/2); i+=THREADS_NUM) {
			double* ptr = posArray + i + j;
                	asm volatile ("{\t\n"
                        	".reg .f64 data;\n\t"
				"ld.global.ca.f64 data, [%1];\n\t"
				"add.f64 %0, data, %0;\n\t"
				"}" : "+d"(sink) : "l"(ptr) : "memory"
                	);
        	}
	}

        // stop timing
        //uint32_t stop = 0;
        //asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	
	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	double *posArray = (double*) malloc(L1_SIZE*sizeof(double));
	double *dsink = (double*) malloc(THREADS_NUM*sizeof(double));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        double *posArray_g;
        double *dsink_g;
	
	for (uint32_t i=0; i<L1_SIZE; i++)
		posArray[i] = (double)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, L1_SIZE*sizeof(double)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(double)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, L1_SIZE*sizeof(double), hipMemcpyHostToDevice) );


	l1_bw<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);
        gpuErrchk( hipPeekAtLastError() );
	
	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(double), hipMemcpyDeviceToHost) );
/*	
	for(uint32_t i=0; i<256; i++){
		printf("stop Clk(%d) = %u    \n", i, stopClk);
		printf("start Clk(%d) = %u    \n", i, startClk);
		printf("Clk(%d) = %u \n", i, stopClk-startClk);
		//printf("dsink(%d) = %f \n", i, dsink);
	}
*/
        double bw;
        bw = (double)(L1_SIZE*L1_SIZE/4*8)/((double)(stopClk[0]-startClk[0]));
        printf("L1 bandwidth = %f (byte/clk)\n", bw);	
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 
